#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "hip/hip_runtime.h"

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "time_cost.hpp"
#include "TernGradEncode_body.h"
#include "get_policy_general.h"
#include "math.h"

struct smaller{
    __host__ __device__
    float operator()(const float& x, const float& y){
        if (x<y){
            return x;
        }
        return y;
    }
};
struct greater{
    __host__ __device__
    float operator()(const float& x, const float& y){
        if (x>y){
            return x;
        }
        return y;
    }
};

// const int N = 1 << 27;
#define N (1<<27)

int main(int argc, char** argv){
    zq_cpp_lib::time_cost zt;
    zt.start();
    // int N = atoi(argv[1]);
    // N = 1 << N;
    int repeat_times = atoi(argv[2]);
    int M = 10 + (N + 3 ) / 4;
    printf("N=%d\n", N);
    printf("M=%d\n", M);
    printf("repeat_times=%d\n", repeat_times);

    // float *devData;
    uint8_t* COMPRESSED;
    hipStream_t stream;
    hiprandGenerator_t gen;
    hipStreamCreate(&stream);
    zt.record("create stream");
    float* hostData;
    hostData = new float[N];
    // hostData = (float *)malloc(N*sizeof(float));
    zt.record("calloc");
    auto policy = zq_cpp_lib::operate_memory::get_policy<thrust::cuda_cub::par_t::stream_attachment_type>::get(stream);
    zt.record("get policy");
    float* devData;
    hipMalloc((void **)&devData, N * sizeof(float));
    // thrust::device_vector<float> devData(N);
    hipMalloc((void **)&COMPRESSED, M * sizeof(uint8_t));
    zt.record("hipMalloc");
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    zt.record("hiprand initialize");
    hiprandGenerateUniform(gen, devData, N);
    hiprandGenerateUniform(gen, devData, N);
    zt.record("generate random twice");
    hipMemcpy(hostData, devData, N * sizeof(float), hipMemcpyDeviceToHost);
    zt.record("hipMemcpy");
    hipStreamSynchronize(stream);

    // float min_value = thrust::reduce(policy, devData, devData+N, 999, thrust::minimum<float>());
    // float max_value = thrust::reduce(policy, devData, devData+N, -99, greater());
    // float max_value = thrust::reduce(thrust::host, hostData, hostData+N, -99, greater());
    // float max_value = thrust::reduce(hostData, hostData+N, -99, greater());

    float data[6] = {1, 0, 2, 2, 1, 3};
    //hostData = {0.8872, 0.5674, 0.4511, 0.4172, 0.0838, 0.7606, ...}
    // hostData = data;

    auto ret_of_max_element = thrust::max_element(thrust::host, hostData, hostData+6);
    printf("ret_of_max_element=%f\n",*ret_of_max_element); //0.8872

    float max_value = thrust::reduce(thrust::host, hostData, hostData+6, -1, thrust::maximum<float>());
    // float max_value = thrust::reduce(thrust::host, hostData, hostData+N, -1, thrust::maximum<float>());
    printf("max_value=%f\n",max_value); //0.00

    float result = thrust::reduce(thrust::host, data, data + 6, -1,thrust::maximum<float>());
    printf("result=%f\n",result); // 3.00

    zt.record("call thrust::reduce");



    for (auto i = 0; i < 16; i++)
    {
        printf("%1.4f ", hostData[i]);
    }
    printf("\n");
    zt.record("print data");
    int ret;
    for (int j = 0; j < repeat_times; j++){
        TernGradEncode_body(
            devData,
            N,
            COMPRESSED,
            M,
            2,
            policy,
            stream
        );
        hipMemcpy(hostData, devData, N * sizeof(float), hipMemcpyDeviceToHost);

        for (auto i = 0; i < 16; i++)
        {
            printf("%1.4f ", hostData[i]);
        }
        printf("\n");
        printf("j=%d\tret=%d\n", j, ret);
    }
    zt.record("call TernGradEncode_body()");
    hiprandDestroyGenerator(gen);
    hipFree(devData);
    hipFree(COMPRESSED);
    zt.record("destroy");
    zt.print_by_us();

    printf("ret=%d\n",ret);

    return 0;
}